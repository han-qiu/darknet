#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "data_type.h"
#include "hip/hip_runtime.h"
#include "convolutional_layer.h"
}
typedef unsigned int uint;
__global__ void trans_kernel(float *x, float *y, int n)
{
#if DATA_TYPE==0
    return;
#elif DATA_TYPE==1
	int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= n) return;
	uint t = *(uint*)&x[i];
	// 1+8+23->1+5+10
	uint frac = t&0x007fffff;
	frac = (frac >> 13)<<13;
	t = (t&(~0x007fffff))|frac;
	y[i] = t;
#else
#error Not done
#endif
}

void trans_gpu(float *x, float *y, int n)
{
    trans_kernel<<<cuda_gridsize(n), BLOCK>>>(x, y, n);
    check_error(hipPeekAtLastError());
}